	
	//
	// cuda event tracking start
	//
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	//
	// cuda event tracking end
	//

	hipEventRecord(stop, 0);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("time: %f \n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
