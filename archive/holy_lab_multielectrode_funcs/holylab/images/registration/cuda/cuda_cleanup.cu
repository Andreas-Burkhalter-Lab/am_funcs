// C++ header
#include <stdio.h>

// cuda header
#include <hip/hip_runtime.h>

// matlab header
#include "mex.h"
#include "matrix.h"

// local header
#include "common/cuda_common.h"

//
// mexFunction: entrance point
//

void mexFunction(int nlhs, mxArray *plhs[], 
							int nrhs, const mxArray *prhs[]) {
	
	//
	// release allocated cuda gpu device
	// execute once after finishing cuda part calculation
	//
	
	hipDeviceReset();
	
}

