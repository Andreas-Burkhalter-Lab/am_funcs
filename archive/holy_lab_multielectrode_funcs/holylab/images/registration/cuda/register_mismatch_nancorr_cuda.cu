#include "hip/hip_runtime.h"
// C++ header
#include <stdio.h>
#include <time.h>

// cuda header
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math_functions.h>
#include <hip/hip_complex.h>

// matlab header
#include "mex.h"
#include "matrix.h"

// local header
#include "common/cuda_common.h"

//
// cuda kernel 1 declaration
//

__global__ void kernel_1( // compute thetaf, fixed, w, wf, wf2, wthetaf
		hipfftDoubleReal *fixed_db,
		hipfftDoubleReal *w_db,
		hipfftReal *fixed, 
		hipfftReal *w,
        hipfftReal *wthetaf, 
        hipfftReal *wf, 
        hipfftReal *wf2,
        int array_length) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	
	while (tid < array_length) {
		float local_fixed = __double2float_rn(fixed_db[tid]);
		float local_w = __double2float_rn(w_db[tid]);
		int thetaf = isnan(local_fixed);
		if (thetaf == 1) local_fixed = 0.0;
		thetaf = !thetaf;
		fixed[tid] = local_fixed;
		w[tid] = local_w;
		wthetaf[tid] = local_w * __int2float_rn(thetaf);
		float local_wf = local_w * local_fixed; 
		wf[tid] = local_wf;
		wf2[tid] = local_wf * local_fixed;
		tid += offset;
	}
}

//
// cuda kernel 11 declaration
//

__global__ void kernel_11( // calculate sum(w(:)) using 1 block of 1024 threads
		hipfftDoubleReal *w_db,
		int array_length,
		hipfftReal *sum_w) {
		
	const uint block_sum_length = 1024;
	__shared__ hipfftReal block_sum[block_sum_length];
	block_sum[threadIdx.x] = 0.0;
	
	int tid = threadIdx.x;
	int offset = blockDim.x;
	while (tid < array_length) {
		block_sum[threadIdx.x] += __double2float_rn(w_db[tid]);
		tid += offset;
	}
	uint i = block_sum_length / 2;
	while (i > blockDim.x) {
		i /= 2;
	}
	while (i != 0) {
		__syncthreads();
		if (threadIdx.x < i)
			block_sum[threadIdx.x] += block_sum[threadIdx.x + i];
		i /= 2;
	}
	if (threadIdx.x == 0)
		sum_w[0] = block_sum[0];
}


//
// cuda kernel 2 declaration
//

__global__ void kernel_2(
		hipfftDoubleReal *moving_db,
		hipfftReal *moving, 
		hipfftReal *thetam,
        hipfftReal *m2, 
        int array_length) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x; // compute moving, thetam, nanflag, m2
	int offset = blockDim.x * gridDim.x;
	while (tid < array_length) {
		float local_moving = __double2float_rn(moving_db[tid]);
		int temp = isnan(local_moving);
		if (temp == 1)
			local_moving = 0.0;
		temp = !temp;
		thetam[tid] = __int2float_rn(temp);
		moving[tid] = local_moving;
		m2[tid] = local_moving * local_moving;
		tid += offset;
	}
}

//
// cuda kernel 3 declaration
//

__global__ void kernel_3(
		hipfftComplex *wf_fft, 
		hipfftComplex *m_fft,
        hipfftComplex *wthetaf_fft, 
        hipfftComplex *m2_fft,
        hipfftComplex *wf2_fft,
        hipfftComplex *thetam_fft,
        hipfftComplex *numerator_fft,
        hipfftComplex *denominator_fft,
        int array_length,
        hipfftReal *sum_w) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x; // compute numerator and denominator before ifftn
	int offset = blockDim.x * gridDim.x;
	while (tid < array_length) {
		hipfftComplex c1 = make_hipFloatComplex(-2.0, 0.0);
		c1 = hipCmulf(c1, hipConjf(wf_fft[tid]));
		c1 = hipCmulf(c1, m_fft[tid]);	
		hipfftComplex c2 = hipCmulf(hipConjf(wthetaf_fft[tid]), m2_fft[tid]);
		c1 = hipCaddf(c1, c2);
		c2 = hipCmulf(hipConjf(wf2_fft[tid]), thetam_fft[tid]);
		numerator_fft[tid] = hipCaddf(c1, c2);	
		
		c1 = hipCmulf(hipConjf(wthetaf_fft[tid]), thetam_fft[tid]);
		c2 = make_hipComplex((1.0 / sum_w[0]), 0.0);
		denominator_fft[tid] = hipCmulf(c1, c2);
		
		tid += offset;
	}
}

//
// cuda kernel_fftshift declaration
//

__global__ void kernel_fftshift(
		hipfftDoubleReal *numerator_db,
		hipfftDoubleReal *denominator_db,
		hipfftReal *numerator,
		hipfftReal *denominator,
		int array_length) {
	
	dim3 offset; // array size along each dimension
	offset.x = gridDim.x * blockDim.x;
	offset.y = gridDim.y * blockDim.y;
	offset.z = gridDim.z * blockDim.z;
	
	dim3 from; // array index: from
	from.x = threadIdx.x + blockDim.x * blockIdx.x;
	from.y = threadIdx.y + blockDim.y * blockIdx.y;
	from.z = threadIdx.z + blockDim.z * blockIdx.z;
	
	dim3 to; // array index: to
	(from.x > (offset.x / 2 - 1)) ? (to.x = from.x - offset.x / 2) : (to.x = from.x + offset.x / 2);
	(from.y > (offset.y / 2 - 1)) ? (to.y = from.y - offset.y / 2) : (to.y = from.y + offset.y / 2);
	(from.z > (offset.z / 2 - 1)) ? (to.z = from.z - offset.z / 2) : (to.z = from.z + offset.z / 2);
	
	int from_tid = static_cast<int> (from.z + from.y * offset.z + from.x * offset.y * offset.z);
	int to_tid = static_cast<int> (to.z + to.y * offset.z + to.x * offset.y * offset.z);
	
	// devide numerator and denominator after ifftn with array_length
	hipfftReal array_length_flt = static_cast<float>(array_length);
	
	numerator_db[to_tid] = static_cast<double>(numerator[from_tid] / array_length_flt);
	denominator_db[to_tid] = static_cast<double>(denominator[from_tid] / array_length_flt);
}



//
// mexFunction
//

void mexFunction(int nlhs, mxArray *plhs[], 
							int nrhs, const mxArray *prhs[]) {

	//
	// check and initialize the input & output arguments
	//

	if (nrhs != 3) // check for proper number of input arguments 
		mexErrMsgTxt("Three input arguments are required.");
	if (nlhs != 2) // check for proper number of output arguments
		mexErrMsgTxt("Two output arguments are required.");

	for (int i = 0; i < nrhs; ++i) {
		if (!mxIsDouble(prhs[i])) // make sure all the input arguments are double
			mexErrMsgTxt("All input arguments must be double.");
	}

	double *h_fixed_db = mxGetPr(prhs[0]); // h_ prefix implies host variable
															        // _db suffix implies double precission
																	// other variables are single precission by default
	mwSize h_fixed_dim = mxGetNumberOfDimensions(prhs[0]);
	const mwSize *h_fixed_size = mxGetDimensions(prhs[0]);

	double *h_moving_db = mxGetPr(prhs[1]);
	mwSize h_moving_dim = mxGetNumberOfDimensions(prhs[1]);
	const mwSize *h_moving_size = mxGetDimensions(prhs[1]);

	double *h_w_db = mxGetPr(prhs[2]);
	mwSize h_w_dim = mxGetNumberOfDimensions(prhs[2]);
	const mwSize *h_w_size = mxGetDimensions(prhs[2]);
	
	for (int i = 0; i < h_fixed_dim; ++i) {
		if ((h_fixed_size[i] != h_moving_size[i])
		        || (h_moving_size[i] != h_w_size[i])
		        || (h_w_size[i] != h_fixed_size[i]))
			mexErrMsgTxt("The input arguments are not of the same size.");
	}

	mwSize plhs_dim = h_fixed_dim; // create output arguments
	const mwSize *plhs_size = h_fixed_size;
	plhs[0] = mxCreateNumericArray(plhs_dim, plhs_size, mxDOUBLE_CLASS, mxREAL);
	plhs[1] = mxCreateNumericArray(plhs_dim, plhs_size, mxDOUBLE_CLASS, mxREAL);
	double *h_numerator_db = mxGetPr(plhs[0]);
	double *h_denominator_db = mxGetPr(plhs[1]);
	
	//
	// define variables related to general array size
	//
	
	int array_dim = static_cast<int>(h_fixed_dim);
	int *array_size = (int *) malloc(array_dim * sizeof(int)); // univeral array size
	int array_length = 1;
	for (int i = 0; i < array_dim; ++i) {
		// switch the array size as matlab & c are using different array storage orders
		array_size[i] = static_cast<int>(h_fixed_size[array_dim - 1 - i]);
		array_length *= array_size[i];
	}
	size_t rBytes = static_cast<size_t>(array_length) * sizeof(hipfftReal); // byte size of real array
	size_t dBytes = rBytes * static_cast<size_t>(2); // byte size of double precission array
	
	//
	// define variables related to "fixed" & "w"
	//
	
	hipfftDoubleReal *d_fixed_db, *d_w_db; // double precission variables
	hipMalloc((void **) &d_fixed_db, dBytes);
	hipMalloc((void **) &d_w_db, dBytes);	
	
	hipfftReal *d_fixed, *d_w, *d_wthetaf, *d_wf, *d_wf2;	// single precission variables
	hipMalloc((void **) &d_fixed, rBytes);
	hipMalloc((void **) &d_w, rBytes);
	hipMalloc((void **) &d_wthetaf, rBytes);
	hipMalloc((void **) &d_wf, rBytes);
	hipMalloc((void **) &d_wf2, rBytes);

	//
	// define variables related to "moving"
	//
	
	hipfftDoubleReal *d_moving_db; // double precission variables
	hipMalloc((void **) &d_moving_db, dBytes);

	hipfftReal *d_moving, *d_thetam, *d_m2; // single precission variables
	hipMalloc((void **) &d_moving, rBytes);
	hipMalloc((void **) &d_thetam, rBytes);
	hipMalloc((void **) &d_m2, rBytes);
	
	//
	// copy double-precission variables from Host to Device
	//
	
	hipMemcpy(d_fixed_db, h_fixed_db, dBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_w_db, h_w_db, dBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_moving_db, h_moving_db, dBytes, hipMemcpyHostToDevice);
	
	//
	// pre-computation related to "fixed" & "w", "moving" before fftn
	//
	
	dim3 blocks, threads, size(array_length, 1, 1);
	cuda_kernel_planning(blocks, threads, size); // optimize these, block size calculator
	
	hipfftReal *d_sum_w;
	hipMalloc((void **) &d_sum_w, sizeof(hipfftReal));
	
	kernel_11<<<1, 1024>>> (
			d_w_db,
			array_length,
			d_sum_w);

	kernel_1<<<blocks, threads>>> (
			d_fixed_db,
			d_w_db,
			d_fixed,
			d_w,
			d_wthetaf,
			d_wf,
			d_wf2,
			array_length);

	kernel_2<<<blocks, threads>>> (
			d_moving_db,
			d_moving,
			d_thetam,
			d_m2,
			array_length);
	
	
	hipDeviceSynchronize(); // make sure all kernels are completed
	cuda_handle_error();
	
	hipFree(d_fixed_db);
	hipFree(d_w_db);
	hipFree(d_moving_db);
	hipFree(d_fixed);
	hipFree(d_w);

	//
	// compute the individual terms for the quadratic
	//

	int array_length_cplx = 1; // complex array length which is less than array_length
	for (int i = 0; i < (array_dim-1); ++i)
		array_length_cplx *= array_size[i];
	array_length_cplx *= (array_size[array_dim-1] / 2 + 1);	
	size_t cBytes = static_cast<size_t>(array_length_cplx) * sizeof(hipfftComplex);

	hipfftComplex *d_wthetaf_fft, *d_m2_fft, *d_wf_fft, *d_m_fft;
	hipMalloc((void**) &d_wthetaf_fft, cBytes);
	hipMalloc((void**) &d_m2_fft, cBytes);
	hipMalloc((void**) &d_wf_fft, cBytes);
	hipMalloc((void**) &d_m_fft, cBytes);

	hipfftComplex *d_wf2_fft, *d_thetam_fft;
	hipMalloc((void **) &d_wf2_fft, cBytes);
	hipMalloc((void **) &d_thetam_fft, cBytes);

	int istride = 1;
	int idist = 0;
	int ostride = 1;
	int odist = 0;
	int batch = 1;

	hipfftHandle plan; // create cufft plan
	hipfftPlanMany(
			&plan, array_dim, array_size, 
			NULL, istride, idist, 
			NULL, ostride, odist, 
			HIPFFT_R2C, batch);
	
	hipfftExecR2C(plan, d_wthetaf, d_wthetaf_fft);
	hipfftExecR2C(plan, d_m2, d_m2_fft);
	hipfftExecR2C(plan, d_wf, d_wf_fft);
	hipfftExecR2C(plan, d_moving, d_m_fft);
	
	hipfftExecR2C(plan, d_wf2, d_wf2_fft);
	hipfftExecR2C(plan, d_thetam, d_thetam_fft);

	hipDeviceSynchronize(); // make sure all FFTN are completed
	cuda_handle_error();
	hipfftDestroy(plan);
	
	hipFree(d_wthetaf);
	hipFree(d_m2);
	hipFree(d_wf);
	hipFree(d_moving);
	
	hipFree(d_wf2);
	hipFree(d_thetam);
	
	//
	// assemble the quadratic and compute ifftn terms
	//

	hipfftComplex *d_numerator_fft, *d_denominator_fft;
	hipMalloc((void **) &d_numerator_fft, cBytes);
	hipMalloc((void **) &d_denominator_fft, cBytes);
	
	kernel_3<<<blocks, threads>>> (
			d_wf_fft, 
			d_m_fft,
	        d_wthetaf_fft, 
	        d_m2_fft,
	        d_wf2_fft,
	        d_thetam_fft,
	        d_numerator_fft,
	        d_denominator_fft,
	        array_length_cplx,
	        d_sum_w);
	
	hipDeviceSynchronize(); // make sure kernel_3 is completed
	cuda_handle_error();
	
	hipFree(d_wthetaf_fft);
	hipFree(d_m2_fft);
	hipFree(d_wf_fft);
	hipFree(d_m_fft);
	hipFree(d_wf2_fft);
	hipFree(d_thetam_fft);
	hipFree(d_sum_w);
	
	hipfftReal *d_numerator, *d_denominator;
	hipMalloc((void **) &d_numerator, rBytes);
	hipMalloc((void **) &d_denominator, rBytes);
	
	hipfftPlanMany(&plan, 
			array_dim, array_size, 
			NULL, istride, idist, 
			NULL, ostride, odist, 
			HIPFFT_C2R, batch);;
	
	hipfftExecC2R(plan, d_numerator_fft, d_numerator);
	hipfftExecC2R(plan, d_denominator_fft, d_denominator);
	
	hipDeviceSynchronize(); // make sure all IFFTN are completed
	cuda_handle_error();
	hipfftDestroy(plan);
	
	hipFree(d_numerator_fft);
	hipFree(d_denominator_fft);
	
	//
	// norminaze denominator & numerator
	//
	
	hipfftDoubleReal *d_numerator_db, *d_denominator_db;
	hipMalloc((void **) &d_numerator_db, dBytes);
	hipMalloc((void **) &d_denominator_db, dBytes);
	
	size.x = static_cast<uint> (array_size[0]);
	size.y = static_cast<uint> (array_size[1]);
	size.z = static_cast<uint> (array_size[2]);
	cuda_kernel_planning2(blocks, threads, size);
	
	kernel_fftshift<<<blocks, threads>>> (
				d_numerator_db,
				d_denominator_db,
				d_numerator,
				d_denominator,
				array_length);
	
	hipDeviceSynchronize();
	hipMemcpy(h_numerator_db, d_numerator_db, dBytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_denominator_db, d_denominator_db, dBytes, hipMemcpyDeviceToHost);
	cuda_handle_error();

	hipFree(d_numerator);
	hipFree(d_denominator);
	hipFree(d_numerator_db);
	hipFree(d_denominator_db);
	
	//
	// release allocated cpu memory
	//
	
	free(array_size);
}



